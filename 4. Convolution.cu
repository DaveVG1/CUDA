
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define N       60000
#define THREADS  1024

// Realiza la convoluci�n secuencial de los valores de los vectores
float convolucionSecuencial(float *vectorA, float *vectorB)
{
	int iPos;
	float fResultado = 0.0;

	// Se multiplican los dos vectores posici�n a posici�n
	for (iPos = 0; iPos < N; iPos++)
		vectorA[iPos] *= vectorB[iPos];

	// Se realiza la convoluci�n
	for (iPos = 0; iPos < N; iPos++)
		fResultado += vectorA[iPos];
	
	return fResultado;
}

__global__ void multParalelaElementoAElemento(float *vectorA, float *vectorB)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < N)
		vectorA[i] *= vectorB[i];
}


// Kernell CUDA para la suma de los valores del vector
__global__ void sumaParalela(float *vector, int n)
{
	__shared__ float vectorCompartido[THREADS];
	
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	// Si el dato est� fuera del vector
	// o si la hebra no tiene que procesar ning�n dato
	if (i >= N || tid >= n)
		vectorCompartido[tid] = 0.0;	// Se rellena con ceros
	else
		vectorCompartido[tid] = vector[i];	// Se copia el dato a la memoria compartida

	__syncthreads();

	for (unsigned int iPos = (blockDim.x >> 1); iPos >= 1; iPos = iPos >> 1)
	{
		if (tid < iPos)
			vectorCompartido[tid] += vectorCompartido[tid + iPos];

		__syncthreads();
	}

	if (tid == 0)
		vector[blockIdx.x] = vectorCompartido[0];
}



int main(void)
{
	float host_vA[N], host_vB[N];
	float fResultadoParalelo, fResultadoSecuencial;
	float *dev_vA, *dev_vB;
	unsigned int blocks;
	unsigned int nDatos;

	// Se llena de forma aleatoria el vector sobre el que se realiza la suma
	srand((unsigned) time(NULL));
	for (int i = 0; i < N; i++)
	{
		host_vA[i] = floorf(10*(rand()/(float)RAND_MAX));
		host_vB[i] = floorf(10*(rand()/(float)RAND_MAX));
	}

	// Pedir memoria en el Device para los vectores a sumar (dev_vA y dev_vB)
	/* COMPLETAR */
	hipMalloc((void **) &dev_vA, N*sizeof(float));
	hipMalloc((void **) &dev_vB, N*sizeof(float));
	// Transferir los vectores del Host al Device
	/* COMPLETAR */
	hipMemcpy(dev_vA, host_vA, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vB, host_vB, N*sizeof(float), hipMemcpyHostToDevice);
	blocks = ceil((float) N / (float) THREADS);

	// Llamada al kernel para hacer la multiplicaci�n elemento a elemento
	/* COMPLETAR */
	multParalelaElementoAElemento <<< N/32, 32 >>>(dev_vA, dev_vB);
	blocks = N;	

	// Llamar al kernell CUDA
	do
	{
		// Se calcula el n�mero de datos que se procesar�n por cada bloque
		if (blocks >= THREADS)
			nDatos = THREADS;
		else
			nDatos = blocks % THREADS;

		// Se calcula el n�mero de bloques necesarios para el n�mero de hebras
		blocks = ceil((float) blocks / (float) THREADS);

		// Llamar al kernel para hacer la resucci�n
		/* COMPLETAR */
		sumaParalela <<< blocks, THREADS >>>(dev_vA, nDatos);
	}
	while (blocks > 1);

	// Copiar el resultado de la operaci�n del Device al Host
	/* COMPLETAR */
	hipMemcpy(&fResultadoParalelo, dev_vA, sizeof(float), hipMemcpyDeviceToHost);
	// Se comprueba que el resultado es correcto y se muestra un mensaje
	fResultadoSecuencial = convolucionSecuencial(host_vA, host_vB);
	if (fResultadoParalelo == fResultadoSecuencial)
		printf("Operacion correcta\nDevice = %f\nHost   = %f\n", fResultadoParalelo, fResultadoSecuencial);
	else
		printf("Operacion INCORRECTA\nDevice = %f\nHost   = %f\n", fResultadoParalelo, fResultadoSecuencial);

	// Librerar la memoria solicitada en el Device
	/* COMPLETAR */
	hipFree(dev_vA);
	hipFree(dev_vB);
	return 0;
}