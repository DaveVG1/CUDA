
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define N  512 

__global__ void add(float *a, float *b,float *c)
{
	int tid = blockIdx.x;

	c[tid] = a[tid] * b[tid];
}

int main(void)
{
	float host_a[N], host_b[N], host_c[N];
	float *dev_a, *dev_b, *dev_c;

	srand((unsigned)time(NULL));
	for (int i= 0;i	<N; i++)
	{
		host_a[i] = floorf(1000 * (rand() / (float)RAND_MAX));
		host_b[i] = floorf(1000 * (rand() / (float)RAND_MAX));
	}

	hipMalloc((void **) &dev_a, N*sizeof(float));
	hipMalloc((void **) &dev_b, N*sizeof(float));
	hipMalloc((void **) &dev_c, N*sizeof(float));

	hipMemcpy(dev_a, host_a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, N*sizeof(float), hipMemcpyHostToDevice);

	add<<<N,1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(host_c, dev_c, N*sizeof(float), hipMemcpyDeviceToHost);

	for (int i= 0;i	<N; i++)
		printf("%f * %f= %f\n", host_a[i], host_b[i], host_c[i]);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}