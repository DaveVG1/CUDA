#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

// Tama�o de la matriz = N * N;
#define N 8192

#define ERROR_CHECK { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
    printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}


// Funci�n secuencial que suma los elementos de dos matrices posici�n a posici�n
void addSec(int *a, int *b, int *c)
{
	// Para cada fila de la matriz
	for (int iFila = 0; iFila < N; iFila++)
		// Para cada columna de la matriz
		for (int iCol = 0; iCol < N; iCol++)
			c[iFila*N+iCol] = a[iFila*N+iCol] + b[iFila*N+iCol];
}

__global__ void addPar(int *a, int *b, int *c)
{
	unsigned int iCol = threadIdx.x + blockDim.x*blockIdx.x;
	unsigned int iFila = threadIdx.y + blockDim.y*blockIdx.y;

	c[iFila*N+iCol] = a[iFila*N+iCol] + b[iFila*N+iCol];
}


int main(void)
{
	int *host_a, *host_b, *host_cSec, *host_cPar;
	int *dev_a, *dev_b, *dev_c;

	// Se pide memoria para las variables del Host
	printf("\nPidiendo memoria en el Host");
	//////////////////////////////////////////////////////////////////////////////////////////////
	//										PASO 1												//
	//									  COMPLETAR												//
	// Pedir memoria para todas las variables en el Host (host_a, host_b, host_cSec, host_cPar) // 
	//////////////////////////////////////////////////////////////////////////////////////////////
	host_a = (int *)malloc(N*N*sizeof(int)); // Host a
	host_b = (int *)malloc(N*N*sizeof(int)); // Host b
	host_cSec = (int *)malloc(N*N*sizeof(int)); // Host c_sec
	host_cPar = (int *)malloc(N*N*sizeof(int)); // Host c_par
	// Se rellenan las matrices con valores aleatorios
	printf("\nRellenando matrices");
	srand((unsigned) time(NULL));
	for (unsigned int i = 0; i < N; i++)
		for (unsigned int j = 0; j < N; j++)
		{
			host_a[i*N+j] = floorf(100*(rand()/(float)RAND_MAX));
			host_b[i*N+j] = floorf(100*(rand()/(float)RAND_MAX));
		}

	// Se reserva memoria en el Device
	printf("\nReservando memoria en el Device");
	//////////////////////////////////////////////////////////////////////////////////////////////
	//										PASO 2												//
	//									  COMPLETAR												//
	// Pedir memoria para todas las variables en el Device (dev_a, dev_b, dev_c)				// 
	//////////////////////////////////////////////////////////////////////////////////////////////
	ERROR_CHECK;
	hipMalloc((void **) &dev_a, N*N*sizeof(int)); // Dev_a
	hipMalloc((void **) &dev_b, N*N*sizeof(int)); // Dev_b
	hipMalloc((void **) &dev_c, N*N*sizeof(int)); // Dev_c
	// Se copian las matrices del Host al Device
	printf("\nCopiando matrices al Device");
	//////////////////////////////////////////////////////////////////////////////////////////////
	//										PASO 3												//
	//									  COMPLETAR												//
	// Copiar las matrices para hacer la suma del Host al Device    							// 
	//////////////////////////////////////////////////////////////////////////////////////////////
	ERROR_CHECK;
	hipMemcpy(dev_a, host_a, N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, N*N*sizeof(int), hipMemcpyHostToDevice);
	//////////////////////////////////////////////////////////////////////////////////////////////
	//										PASO 4												//
	//									  COMPLETAR												//
	// Preparar la llamada al kernel y llamarlo con los par�metros adecuados    				// 
	//////////////////////////////////////////////////////////////////////////////////////////////
	dim3 gridOfBlocks(N/32,N/32); // Cada Warp suelta 32 hebras, funcionamiento �ptimo
	dim3 blockOfThreads(32,32); // Cuadra con el n�mero de hebras por bloque

	printf("\nSuma paralela");
	addPar<<<gridOfBlocks,blockOfThreads>>> (dev_a, dev_b, dev_c);
	ERROR_CHECK;
	// Se copia la matriz resultado del Device al Host
	printf("\nCopiando matriz resultado al Host");
	//////////////////////////////////////////////////////////////////////////////////////////////
	//										PASO 5												//
	//									  COMPLETAR												//
	// Copiar la matriz resultado del Device al Host			    							// 
	//////////////////////////////////////////////////////////////////////////////////////////////
	hipMemcpy(host_cPar, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
	// Se llama a la suma de matrices secuencial
	printf("\nSuma secuencial");
	addSec(host_a, host_b, host_cSec);

	// Se compara si las matrices resultado paralela y secuencial son iguales
	printf("\nComprobando resultados");
	bool bError = false;
	for (int iFila = 0; iFila < N; iFila++)
	{
		for (int iCol = 0; iCol < N; iCol++)
  			if (host_cSec[iFila*N+iCol] != host_cPar[iFila*N+iCol])
			{
				printf("\nValores diferentes en [%d][%d] => (%d,%d) => %d %d", iCol, iFila, host_a[iFila*N+iCol], host_b[iFila*N+iCol], host_cSec[iFila*N+iCol], host_cPar[iFila*N+iCol]);
				bError = true;
				break;
			}
		if (bError)
			break;
	}
	if (!bError)
		printf("\nCORRECTO!\n");

	// Se libera la memoria del Host
	//////////////////////////////////////////////////////////////////////////////////////////////
	//										PASO 6												//
	//									  COMPLETAR												//
	// Liberar la memoria pedida en el Host			    										// 
	//////////////////////////////////////////////////////////////////////////////////////////////
	free(host_a);
	free(host_b);
	free(host_cSec);
	free(host_cPar);
	// Se libera la memoria del Device
	//////////////////////////////////////////////////////////////////////////////////////////////
	//										PASO 7												//
	//									  COMPLETAR												//
	// Liberar la memoria pedida en el Device		    										// 
	//////////////////////////////////////////////////////////////////////////////////////////////
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}